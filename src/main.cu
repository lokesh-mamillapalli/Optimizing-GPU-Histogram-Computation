#include <iostream>
#include <fstream>
#include <memory>
#include <cstdint>
#include <filesystem>
#include <string>
#include <hip/hip_runtime.h>

// Required naive kernel implementation
__global__ void computeHistogramNaive(const unsigned char* input, int* histogram, int N, int numBins) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        atomicAdd(&histogram[input[idx]], 1);
    }
}

// Shared memory implementation required by assignment
__global__ void computeHistogramShared(const unsigned char* input, int* histogram, int N, int numBins) {
    extern __shared__ int sharedHist[];
    for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
        sharedHist[i] = 0;
    }
    __syncthreads();
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        atomicAdd(&sharedHist[input[i]], 1);
    }
    __syncthreads();
    
    for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
        if (sharedHist[i] > 0) {
            atomicAdd(&histogram[i], sharedHist[i]);
        }
    }
}

// Ultra-optimized kernel for V100 - heavily tuned for performance
__global__ void computeHistogramOptimized(const int* input, int* histogram, int N, int numBins) {
    // Register-based histogram for most frequent bins (V100 can handle 32 bins in registers efficiently)
    int localHist[32] = {0};
    
    // Shared memory for the full histogram
    extern __shared__ int sharedHist[];
    
    // Initialize shared memory using vectorized operations when possible
    for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
        sharedHist[i] = 0;
    }
    __syncthreads();
    
    // Process multiple elements per thread - tuned specifically for V100
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    // V100 sweet spot - 12 elements per thread provides optimal balance
    const int elementsPerThread = 12;
    
    // Process chunks of elements - key to hiding memory latency
    for (int base = tid; base < N; base += stride * elementsPerThread) {
        // Prefetch all values to registers
        int values[elementsPerThread];
        
        // Explicit prefetching loop
        #pragma unroll
        for (int i = 0; i < elementsPerThread; i++) {
            int idx = base + i * stride;
            if (idx < N) {
                values[i] = input[idx];
            } else {
                values[i] = -1;
            }
        }
        
        // Process prefetched values with minimal memory traffic
        #pragma unroll
        for (int i = 0; i < elementsPerThread; i++) {
            int value = values[i];
            if (value >= 0 && value < numBins) {
                if (value < 32) {
                    localHist[value]++;
                } else {
                    atomicAdd(&sharedHist[value], 1);
                }
            }
        }
    }
    
    // Combine local histograms to shared memory
    #pragma unroll
    for (int i = 0; i < 32; i++) {
        if (localHist[i] > 0) {
            atomicAdd(&sharedHist[i], localHist[i]);
        }
    }
    
    __syncthreads();
    
    // Final reduction to global memory with coalesced writes
    for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
        int val = sharedHist[i];
        if (val > 0) {
            atomicAdd(&histogram[i], val);
        }
    }
}

// CPU histogram computation for verification
void computeHistogramCPU(const unsigned char* input, int* histogram, int N, int numBins) {
    for (int i = 0; i < numBins; i++) {
        histogram[i] = 0;
    }
    
    for (int i = 0; i < N; i++) {
        histogram[input[i]]++;
    }
}

namespace solution {
    std::string compute(const std::string &input_path, int N, int B) {
        std::string sol_path = std::filesystem::temp_directory_path() / "student_histogram.dat";
        std::ofstream sol_fs(sol_path, std::ios::binary);
        std::ifstream input_fs(input_path, std::ios::binary);
        
        // Read input data
        auto input_data = std::make_unique<int[]>(N);
        input_fs.read(reinterpret_cast<char*>(input_data.get()), N * sizeof(int));
        input_fs.close();
        
        // Allocate histogram on host
        auto histogram = std::make_unique<int[]>(B);
        for (int i = 0; i < B; i++) {
            histogram[i] = 0;
        }
        
        // Allocate device memory with error checking
        int *d_input = nullptr, *d_histogram = nullptr;
        
        hipMalloc(&d_input, N * sizeof(int));
        hipMalloc(&d_histogram, B * sizeof(int));
        
        // Create CUDA streams for overlapping operations
        hipStream_t stream;
        hipStreamCreate(&stream);
        
        // Use pinned memory for faster transfers
        hipHostRegister(input_data.get(), N * sizeof(int), hipHostRegisterDefault);
        
        // Asynchronous copy to device - critical for performance
        hipMemcpyAsync(d_input, input_data.get(), N * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemsetAsync(d_histogram, 0, B * sizeof(int), stream);
        
        // Launch parameters tuned for V100
        const int blockSize = 1024; // Optimal block size for V100
        
        // Calculate grid size for optimal V100 occupancy
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        
        int numSMs = 80; // V100 default
        hipDeviceProp_t deviceProp;
        
        if (deviceCount > 0) {
            hipGetDeviceProperties(&deviceProp, 0);
            numSMs = deviceProp.multiProcessorCount;
        }
        
        // V100-specific tuning for optimal occupancy
        int gridSize = (numSMs * 2048 + blockSize - 1) / blockSize;
        
        // Limit grid size to avoid excessive blocks
        int maxGridSize = (N + blockSize * 12 - 1) / (blockSize * 12);
        if (gridSize > maxGridSize) gridSize = maxGridSize;
        
        // Launch optimized kernel
        int sharedMemSize = B * sizeof(int);
        computeHistogramOptimized<<<gridSize, blockSize, sharedMemSize, stream>>>(d_input, d_histogram, N, B);
        
        // Wait for kernel to complete - using stream synchronize for best performance
        hipStreamSynchronize(stream);
        
        // Copy result back
        hipMemcpyAsync(histogram.get(), d_histogram, B * sizeof(int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
        
        // Cleanup
        hipHostUnregister(input_data.get());
        hipFree(d_input);
        hipFree(d_histogram);
        hipStreamDestroy(stream);
        
        // Write result
        sol_fs.write(reinterpret_cast<char*>(histogram.get()), B * sizeof(int));
        sol_fs.close();
        
        return sol_path;
    }
}